/* Enunciado:
 * Multiplicacion de Matrices MxN (16x16) por Bloques en CUDA
 */
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

hipError_t prodMatricesCuda(int *c, const int *a, const int *b, unsigned int Width);
const int TILE_WIDTH = 4;//Se ha establecido un tama�o de tesela de 4 hilos

__global__ void productoKernel(int *c, const int *a, const int *b, unsigned int Width)
{
	int id_fil = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int id_col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int n = 0;

	for (int i = 0; i < Width; i++) {
		n = n + (a[id_fil*Width + i] * b[i*Width + id_col]);
	}
    c[id_fil*Width + id_col] = n;
}

void imprimeMatriz(int *v, int m, int n) {//( m * n )
	int i, j, x;
	int ws;//numero de espacios de caracteres por casilla
	printf("\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			ws = 5;
			x = v[i*m + j];

			if (x < 0) {//si es negativo, se ocupa un hueco por el signo "-"
				ws--;
				x = -1 * x;
			}
			else {//para alinear los d�gitos
				ws--;
				printf(" ");
			}
			do {//Se ocupa un hueco por digito del numero
				ws--;
				x = x / 10;
			} while (x > 0);

			printf("%d", v[i*m + j]);//imprimimos el numero
			while (ws > 0) {//y ocupamos el resto de huecos con espacios en blanco
				printf(" ");
				ws--;
			}
		}
		printf("\n");
	}
}

void imprimeMatriz(int *v, int m) {//Para matrices cuadradas ( m * m )
	int i, j, x;
	int ws;//numero de espacios de caracteres por casilla
	printf("\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < m; j++) {
			ws = 5;
			x = v[i*m + j];

			if (x < 0) {//si es negativo, se ocupa un hueco por el signo "-"
				ws--;
				x = -1 * x;
			}
			else {//para alinear los d�gitos
				ws--;
				printf(" ");
			}
			do {//Se ocupa un hueco por digito del numero
				ws--;
				x = x / 10;
			} while (x > 0);

			printf("%d", v[i*m + j]);//imprimimos el numero
			while (ws > 0) {//y ocupamos el resto de huecos con espacios en blanco
				printf(" ");
				ws--;
			}
		}
		printf("\n");
	}
}

void generaMatriz(int *v, int m, int n, int max, int min) {//( m * n )
	int i, j;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			v[i*n + j] = (rand() % (max - min)) + min;
		}
	}
}

void generaMatriz(int *v, int m, int max, int min) {//Para matrices cuadradas ( m * m )
	int i, j;
	for (i = 0; i < m; i++) {
		for (j = 0; j < m; j++) {
			v[i*m + j] = (rand() % (max - min)) + min;
		}
	}
}

int main()
{
	srand(time(0));
	const int max = 10;
	const int min = -10;
    const int Width = 16;
	int a[Width * Width] = { 0 };
	generaMatriz(a, Width, max, min);

	int b[Width * Width] = { 0 };
	generaMatriz(b, Width, max, min);

    int c[Width * Width] = { 0 };
	
    // Add vectors in parallel.
    hipError_t cudaStatus = prodMatricesCuda(c, a, b, Width);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	printf("\n\tMatriz A\n");
	imprimeMatriz(a, Width);
	printf("\n\tProducto de Matriz B\n");
	imprimeMatriz(b, Width);
	printf("\n\tResultado:\n");
	imprimeMatriz(c, Width);
	
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t prodMatricesCuda(int *c, const int *a, const int *b, unsigned int Width)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
	dim3 DimGrid(Width/TILE_WIDTH, Width / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, Width * Width * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, Width * Width * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, Width * Width * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, Width * Width * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, Width * Width * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    productoKernel<<<DimGrid, DimBlock>>>(dev_c, dev_a, dev_b, Width);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, Width * Width * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
